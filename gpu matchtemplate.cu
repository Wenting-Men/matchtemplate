#include "hip/hip_runtime.h"


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <iostream>
#include <opencv2\opencv.hpp>
#include "highgui.h"
#include <stdlib.h>
#include <math.h>
#include <Windows.h>

using namespace std;
using namespace cv;


#define VALUE_MAX 1000000

struct match 
{

  int diffRow;
	int diffCol;
	int diff;

} pos;

// 定义CPU函数
void CPU_ImgFindDiff(int *host_result,int Width,int Height, int tWidth, int tHeight);
// 定义GPU
__global__ void GPU_Kernel_ImgMatching(unsigned char * d_ImgSrc ,unsigned char *d_pImgSrc,int *d_diffDst,int Width,int Height, int tWidth, int tHeight);


int main(int argc, char *argv[])
{
	
  //开始计时
  DWORD start_time = GetTickCount();     

	Mat srcImg = imread("C:\\Users\\xx.jpg", 0);
	Mat temImg = imread("C:\\Users\\template.jpg", 0);
  //定义宽高
  int Width = srcImg.cols;
  int Height = srcImg.rows;
	int tWidth = temImg.cols;
  int tHeight = temImg.rows;
  // 定义大小
	size_t d_sizeDiff = sizeof(int) * (Width - tWidth + 1) * (Height - tHeight + 1) ;
	size_t d_sizeImg  = sizeof(unsigned char) * Width * Height;
	size_t d_psizeImg = sizeof(unsigned char) * tWidth * tHeight;
	
  // CPU内存设置
	unsigned char *h_ImgSrc  = (unsigned char*)(srcImg.data);
	unsigned char *h_pImgSrc = (unsigned char*)(temImg.data);
	int *h_diffDst = (int *)malloc(d_sizeDiff);

  // GPU内存设置
	int *d_diffDst 			 = NULL;
	unsigned char *d_ImgSrc  = NULL;
	unsigned char *d_pImgSrc = NULL;

	hipMalloc((void**)&d_diffDst, d_sizeDiff);
	hipMalloc((void**)&d_ImgSrc , d_sizeImg);
	hipMalloc((void**)&d_pImgSrc, d_psizeImg);

	hipMemcpy(d_diffDst, h_diffDst, d_sizeDiff, hipMemcpyHostToDevice);
	hipMemcpy(d_pImgSrc, h_pImgSrc, d_psizeImg, hipMemcpyHostToDevice);
	hipMemcpy(d_ImgSrc , h_ImgSrc , d_sizeImg , hipMemcpyHostToDevice);
	// 定义 block 和 thread
	dim3 threads(32);
  dim3 grid(256,256);
	// 调用kernel函数
  GPU_Kernel_ImgMatching<<<grid, threads>>>(d_ImgSrc, d_pImgSrc, d_diffDst, Width, Height, tWidth, tHeight);
  // 传回cpu 
  hipMemcpy(h_diffDst, d_diffDst, d_sizeDiff, hipMemcpyDeviceToHost);
  //找点

  DWORD end_time1 = GetTickCount();      
  cout << "gpu：" << (end_time1 - start_time) << "ms!" << endl;  
  CPU_ImgFindDiff( h_diffDst,Width, Height, tWidth, tHeight);
  CvPoint pt1, pt2;

	pt1.x = pos.diffCol;
	pt1.y = pos.diffRow;
	pt2.x = pt1.x + temImg.cols;
	pt2.y = pt1.y + temImg.rows;	
  
  
	DWORD end_time2 = GetTickCount();
	cout << "找点：" << (end_time2 - end_time1) << "ms!" << endl;  
	
  // 画图
	rectangle( srcImg, pt1, pt2, CV_RGB(255,0,0), 3, 8, 0 );
  imshow( "result", srcImg );


	//释放内存
	hipFree(d_diffDst);
	hipFree(d_pImgSrc);
	hipFree(d_ImgSrc);

	DWORD end_time3 = GetTickCount();       
	cout << "全体：" << (end_time3 - start_time) << "ms!" << endl;  
	waitKey(0); 
	return 0;

}



__global__ void GPU_Kernel_ImgMatching(unsigned char * d_ImgSrc ,unsigned char *d_pImgSrc,int *d_diffDst,int Width,int Height, int tWidth, int tHeight)
{
	     
 	int diff;
 	int result_height = Height - tHeight + 1;
	int result_width  = Width  - tWidth  + 1;
   
  uchar p_srcImg, p_temImg;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  	
  if(tid < Width ) 
  {
    for(int row = 0; row < result_height; row++ ) 
    {
      diff = 0;
			for(int i=0; i<tHeight; i++) 
      {            
         for(int j=0; j<tWidth; j++) 
         {    
					 p_srcImg = d_ImgSrc[(row + i) * Width + tid + j];
					 p_temImg = d_pImgSrc[i * tWidth + j];
					 diff += fabsf(p_srcImg - p_temImg);
					 //printf("debug:%d\n",diff);//利用输出debug
				  }       
    	} 
    	d_diffDst[row * result_width + tid] = diff;
    }
  }
}


void CPU_ImgFindDiff( int *host_result,int Width,int Height, int tWidth, int tHeight)
{
	
	int minDiff = VALUE_MAX;
	int result_height = Height - tHeight + 1;
	int result_width  = Width  - tWidth  + 1;

	for( int row = 0; row < result_height; row++ ) 
  {
		for( int col = 0; col < result_width; col++ ) 
    {
			if ( minDiff > host_result[row * result_width + col] ) 
      {
				minDiff = host_result[row * result_width + col];

				pos.diffRow = row;
				pos.diffCol = col;
				pos.diff = host_result[row * result_width + col];
			}
		}
	}

	//printf("minSAD:%d\n",minDiff);

	free(host_result);
}
